#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""


#include "hip/hip_runtime.h"

static const int THREADS = 28;
static const int SLICES = 1;
__global__ void grey_scale(char* imageData, int rows, int columns, int channels) {
	int thread_id = threadIdx.x;
	int from = rows/THREADS * thread_id;
	int to = rows/THREADS * (thread_id + 1);

	for (int x = from; x < to; x++) {
		for (int y = 0; y < columns; y++) {
			auto rgb = imageData[x * columns * channels + y * channels] * 0.3f;
			rgb += imageData[x * columns * channels + y * channels + 1] * 0.59f;
			rgb += imageData[x * columns * channels + y * channels + 2] * 0.11f;
			imageData[x * columns * channels + y * channels] = (char)(rgb);
			imageData[x * columns * channels + y * channels + 1] = (char)(imageData[x * columns * channels + y * channels]);
			imageData[x * columns * channels + y * channels + 2] = (char)(imageData[x * columns * channels + y * channels]);
		}
	}
}

void image_cuda(char *imageData, size_t size, int rows, int cols, int channels) {
	char *dev_image;

	hipMalloc((void **)&dev_image, size);
	hipMemcpy(dev_image, imageData, size, hipMemcpyHostToDevice);

	grey_scale << < SLICES, THREADS >> > (dev_image, rows, cols, channels);

	hipMemcpy(imageData, dev_image, size, hipMemcpyDeviceToHost);
	hipFree(dev_image);
}